#include "hip/hip_runtime.h"
/*
 *
 * Author: Shuojiang Liu
 * Class: ECE6122
 * Last Date Modified: November 6, 2023
 * Description: This program will implement a CUDA program to simulate a 2D random walk.
 * A random walk is a mathematical process that describes a path consisting of a sequence of random steps.
 * Simulate a large number of walkers taking steps either north, south, east, or west on a grid,
 * and calculate the average distance they travel from the origin.
 */

#include <cmath>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include "error.cuh"

#define BLOCK_SIZE 256

// GPU function to perform a random walk simulation
__global__ void randomWalkMethod(int *x, int *y, int numSteps, int numWalkers)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id >= numWalkers)
    {
        return;
    }

    hiprandState state;
    // we use thread index as the seed
    hiprand_init(threadIdx.x + blockIdx.x * blockDim.x, id, 0, &state);

    for (int step = 0; step < numSteps; step++)
    {
        float randomNumber = hiprand_uniform(&state);
        if (randomNumber < 0.25f)
            x[id] += 1;
        else if (randomNumber < 0.5f)
            x[id] -= 1;
        else if (randomNumber < 0.75f)
            y[id] += 1;
        else
            y[id] -= 1;
    }
}

// CPU function to calculate the average distance from the origin
float averageDistance(const int *x, const int *y, int numWalkers)
{
    float sumDistance = 0.0f;
    for (int i = 0; i < numWalkers; i++)
    {
        float distance = sqrtf(x[i] * x[i] + y[i] * y[i]);
        sumDistance += distance;
    }
    return sumDistance / static_cast<float>(numWalkers);
}

// Use Normal CUDA memory Allocation
void simulationNormal(int numWalkers, int numSteps)
{
    int* d_x, * d_y, * h_x, * h_y;
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    CHECK(hipMalloc(&d_x, numWalkers * sizeof(int)));
    CHECK(hipMalloc(&d_y, numWalkers * sizeof(int)));
    h_x = new int[numWalkers];
    h_y = new int[numWalkers];

    CHECK(hipEventRecord(start));

    int numBlocks = (numWalkers + BLOCK_SIZE - 1) / BLOCK_SIZE;
    randomWalkMethod<<<numBlocks, BLOCK_SIZE>>>(d_x, d_y, numSteps, numWalkers);

    CHECK(hipDeviceSynchronize());

    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));

    CHECK(hipMemcpy(h_x, d_x, numWalkers * sizeof(int), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_y, d_y, numWalkers * sizeof(int), hipMemcpyDeviceToHost));

    float milliseconds = 0;
    CHECK(hipEventElapsedTime(&milliseconds, start, stop));

    float averageDist = averageDistance(h_x, h_y, numWalkers);

    std::cout << "Normal CUDA memory Allocation:" << std::endl;
    std::cout << "    Time to calculate(microsec): " << milliseconds * 1000 << std::endl;
    std::cout << "    Average distance from origin: " << averageDist << std::endl;

    CHECK(hipFree(d_x));
    CHECK(hipFree(d_y));

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
}


// Use Pinned CUDA memory Allocation
void simulationPinned(int numWalkers, int numSteps)
{
    int *d_x, *d_y, *h_x, *h_y;
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    CHECK(hipHostMalloc(&h_x, numWalkers * sizeof(int)));
    CHECK(hipHostMalloc(&h_y, numWalkers * sizeof(int)));

    CHECK(hipMalloc(&d_x, numWalkers * sizeof(int)));
    CHECK(hipMalloc(&d_y, numWalkers * sizeof(int)));

    CHECK(hipEventRecord(start));

    int numBlocks = (numWalkers + BLOCK_SIZE - 1) / BLOCK_SIZE;
    randomWalkMethod<<<numBlocks, BLOCK_SIZE>>>(d_x, d_y, numSteps, numWalkers);

    CHECK(hipDeviceSynchronize());

    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));

    CHECK(hipMemcpy(h_x, d_x, numWalkers * sizeof(int), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_y, d_y, numWalkers * sizeof(int), hipMemcpyDeviceToHost));

    float milliseconds = 0;
    CHECK(hipEventElapsedTime(&milliseconds, start, stop));

    float averageDist = averageDistance(h_x, h_y, numWalkers);

    std::cout << "Pinned CUDA memory Allocation:" << std::endl;
    std::cout << "    Time to calculate(microsec): " << milliseconds * 1000 << std::endl;
    std::cout << "    Average distance from origin: " << averageDist << std::endl;

    CHECK(hipFree(d_x));
    CHECK(hipFree(d_y));
    CHECK(hipHostFree(h_x));
    CHECK(hipHostFree(h_y));

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
}

// Use Managed CUDA memory Allocation
void simulationManaged(int numWalkers, int numSteps)
{
    int *d_x, *d_y;
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    CHECK(hipMallocManaged(&d_x, numWalkers * sizeof(int)));
    CHECK(hipMallocManaged(&d_y, numWalkers * sizeof(int)));

    CHECK(hipEventRecord(start));

    int numBlocks = (numWalkers + BLOCK_SIZE - 1) / BLOCK_SIZE;
    randomWalkMethod<<<numBlocks, BLOCK_SIZE>>>(d_x, d_y, numSteps, numWalkers);

    CHECK(hipDeviceSynchronize());

    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));

    float milliseconds = 0;
    CHECK(hipEventElapsedTime(&milliseconds, start, stop));

    float averageDist = averageDistance(d_x, d_y, numWalkers);

    std::cout << "Managed CUDA memory Allocation:" << std::endl;
    std::cout << "    Time to calculate(microsec): " << milliseconds * 1000 << std::endl;
    std::cout << "    Average distance from origin: " << averageDist << std::endl;

    CHECK(hipFree(d_x));
    CHECK(hipFree(d_y));

    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
}

/*
Takes as program input arguments the Number of Walkers,
and the number of steps each walker needs to take on a 2D integer grid.
Use command line flags to distinguish Number Walkers (-W) and (-I) for number of steps.
All the walkers start at the origin (0, 0).
*/
int main(int argc, char *argv[])
{
    int numWalkers = 1000;
    int numSteps = 10000;

    if (argc > 1)
    {
        for (int i = 1; i < argc; i++)
        {
            if (strcmp(argv[i], "-W") == 0)
            {
                numWalkers = atoi(argv[++i]);
            }
            else if (strcmp(argv[i], "-I") == 0)
            {
                numSteps = atoi(argv[++i]);
            }
        }
    }

    simulationNormal(numWalkers, numSteps);
    simulationPinned(numWalkers, numSteps);
    simulationManaged(numWalkers, numSteps);

    std::cout << "Bye" << std::endl;

    return 0;
}
